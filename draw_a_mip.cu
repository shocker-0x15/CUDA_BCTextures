#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

extern "C" __global__ void drawMip(
	hipTextureObject_t texture, int32_t mipLevel,
	uint32_t* dstImg, int2 dstImgSize) {
	const int2 pixIdx(
		blockDim.x * blockIdx.x + threadIdx.x,
		blockDim.y * blockIdx.y + threadIdx.y);
	float4 texValue = tex2DLod<float4>(
		texture, (pixIdx.x + 0.5f) / dstImgSize.x, (pixIdx.y + 0.5f) / dstImgSize.y, mipLevel);
	//printf("%4u, %4u: (%g, %g, %g, %g)\n", pixIdx.x, pixIdx.y, texValue.x, texValue.y, texValue.z, texValue.w);
	dstImg[pixIdx.y * dstImgSize.x + pixIdx.x] =
		(min(static_cast<uint32_t>(255 * texValue.x), 255u) << 0)
		| (min(static_cast<uint32_t>(255 * texValue.y), 255u) << 8)
		| (min(static_cast<uint32_t>(255 * texValue.z), 255u) << 16)
		| (min(static_cast<uint32_t>(255 * texValue.w), 255u) << 24);
}
